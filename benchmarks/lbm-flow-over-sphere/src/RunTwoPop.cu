#include "hip/hip_runtime.h"
#include "Config.h"
#include "D3Q19.h"
#include "Neon/domain/dGrid.h"

#include "CellType.h"
#include "LbmSkeleton.h"
#include "Metrics.h"
#include "Repoert.h"
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif
#include <fenv.h>
namespace CavityTwoPop {

int backendWasReported = false;

namespace details {
template <typename Grid,
          typename StorageFP,
          typename ComputeFP>
auto runSpecialized(Config& config,
                    Report& report) -> void
{
    using Lattice = D3Q19Template<StorageFP, ComputeFP>;
    using PopulationField = typename Grid::template Field<StorageFP, Lattice::Q>;

    feenableexcept(FE_DIVBYZERO);

    Neon::Backend bk = [&] {
        if (config.deviceType == "cpu") {
            Neon::Backend bk(config.devices, Neon::Runtime::openmp);
            return bk;
        }
        if (config.deviceType == "gpu") {
            Neon::Backend bk(config.devices, Neon::Runtime::stream);
            return bk;
        }
        Neon::NeonException exce("run");
        exce << config.deviceType << " is not a supported option as device type";
        NEON_THROW(exce);
    }();

    if (!backendWasReported) {
        metrics::recordBackend(bk, report);
        backendWasReported = true;
    }

    Lattice               lattice(bk);
    const double          radiusDomainLenRatio = 1.0 / 7;
    const Neon::double_3d center = {config.N / 2.0, config.N / 2.0, config.N / 2.0};
    const double          radius = config.N * radiusDomainLenRatio;
    const double          rhoPrescribedInlet = 1.0;
    const double          rhoPrescribedOutlet = 1.005;

    auto isFluidDomain =
        [&](const Neon::index_3d& idx)
        -> bool {
        if (idx < 0)
            return false;
        if (idx.x >= config.N ||
            idx.y >= config.N ||
            idx.z >= config.N) {
            return false;
        }
        const auto point = idx.newType<double>();
        const auto offset = std::pow(point.x - center.x, 2) +
                            std::pow(point.y - center.y, 2) +
                            std::pow(point.z - center.z, 2);
        if (offset <= radius * radius) {
            // we are in the sphere
            return false;
        }
        return true;
    };

    auto isInsideSphere =
        [&](const Neon::index_3d& idx) -> bool {
        if (idx.x < 0 ||
            idx.y < 0 ||
            idx.z < 0)
            return false;
        if (idx.x >= config.N ||
            idx.y >= config.N ||
            idx.z >= config.N) {
            return false;
        }
        const auto point = idx.newType<double>();
        const auto offset = std::pow(point.x - center.x, 2) +
                            std::pow(point.y - center.y, 2) +
                            std::pow(point.z - center.z, 2);
        if (offset <= radius * radius) {
            // we are in the sphere
            return true;
        }
        return false;
    };

    auto getBoundaryType =
        [&](const Neon::index_3d& idx) -> CellType::Classification {
        if (idx.z == 0 || idx.z == config.N - 1) {
            return CellType::Classification::bounceBack;
        }
        if (idx.y == 0 || idx.y == config.N - 1) {
            return CellType::Classification::bounceBack;
        }
        if (idx.x == 0 || idx.x == config.N - 1) {
            return CellType::Classification::bounceBack;
        }

        auto idEdge = [idx, config](int d1, int d2) {
            if ((idx.v[d1] == 1 && idx.v[d2] == 1) ||
                (idx.v[d1] == 1 && idx.v[d2] == config.N - 2) ||
                (idx.v[d1] == config.N - 2 && idx.v[d2] == 1) ||
                (idx.v[d1] == config.N - 2 && idx.v[d2] == config.N - 2)) {
                return true;
            }
            return false;
        };

        if (idEdge(0,1)) {
            return CellType::Classification::bulk;
        }
        if (idEdge(0,2)) {
            return CellType::Classification::bulk;
        }
        if (idEdge(1,2)) {
            return CellType::Classification::bulk;
        }

        if (idx.x == 1) {
            return CellType::Classification::pressure;
        }
        if (idx.x == config.N - 2) {
            return CellType::Classification::velocity;
        }
        if (isInsideSphere(idx)) {
            return CellType::Classification::undefined;
        }
        for (int i = -1; i < 2; i++) {
            for (int j = -1; j < 2; j++) {
                for (int k = -1; k < 2; k++) {
                    Neon::index_3d offset(i, j, k);
                    Neon::index_3d neighbour = idx + offset;
                    bool           isIn = isInsideSphere(neighbour);
                    if (isIn) {
                        return CellType::Classification::bounceBack;
                    }
                }
            }
        }
        return CellType::Classification::bulk;
    };

    // Neon Grid and Fields initialization
    auto [start, clock_iter] = metrics::restartClock(bk, true);
    Grid grid(
        bk,
        {config.N, config.N, config.N},
        [](const Neon::index_3d&) { return true; },
        lattice.c_vect);

    PopulationField pop0 = grid.template newField<StorageFP, Lattice::Q>("Population", Lattice::Q, StorageFP(0.0));
    PopulationField pop1 = grid.template newField<StorageFP, Lattice::Q>("Population", Lattice::Q, StorageFP(0.0));

    typename Grid::template Field<StorageFP, 1> rho;
    typename Grid::template Field<StorageFP, 3> u;

    if (!config.benchmark) {
        std::cout << "Allocating rho and u" << std::endl;
        rho = grid.template newField<StorageFP, 1>("rho", 1, StorageFP(0.0));
        u = grid.template newField<StorageFP, 3>("u", 3, StorageFP(0.0));
    }


    CellType defaultCelltype;
    auto     flag = grid.template newField<CellType, 1>("Material", 1, defaultCelltype);
    auto     bcTypeForDebugging = grid.template newField<double, 1>("BCtype", 1, 33);

    auto lbmParameters = config.getLbmParameters<ComputeFP>();

    LbmIterationD3Q19<PopulationField, ComputeFP>
        iteration(config.stencilSemantic,
                  config.occ,
                  config.transferMode,
                  pop0,
                  pop1,
                  flag,
                  lbmParameters.omega);

    auto exportRhoAndU = [&bk, &rho, &u, &iteration, &flag](int iterationId) {
        if (true) {
            std::cout << "Exporting\n"
                      << std::endl;
            auto& f = iteration.getInput();
            bk.syncAll();
            Neon::set::HuOptions hu(Neon::set::TransferMode::get,
                                    false,
                                    Neon::Backend::mainStreamIdx,
                                    Neon::set::StencilSemantic::standard);

            f.haloUpdate(hu);
            bk.syncAll();
            auto container = LbmContainers<Lattice, PopulationField, ComputeFP>::computeRhoAndU(f, flag, rho, u);

            container.run(Neon::Backend::mainStreamIdx);
            u.updateIO(Neon::Backend::mainStreamIdx);
            rho.updateIO(Neon::Backend::mainStreamIdx);
            // iteration.getInput().updateIO(Neon::Backend::mainStreamIdx);

            bk.syncAll();
            size_t      numDigits = 5;
            std::string iterIdStr = std::to_string(iterationId);
            iterIdStr = std::string(numDigits - std::min(numDigits, iterIdStr.length()), '0') + iterIdStr;

            u.ioToVtk("u_" + iterIdStr, "u", false);
            rho.ioToVtk("rho_" + iterIdStr, "rho", false);
            // iteration.getInput().ioToVtk("pop_" + iterIdStr, "u", false);
            // flag.ioToVtk("flag_" + iterIdStr, "u", false);
        }
    };


    metrics::recordGridInitMetrics(bk, report, start);
    tie(start, clock_iter) = metrics::restartClock(bk, true);

    // Problem Setup
    // 1. init all lattice to equilibrium
    {
        auto& inPop = iteration.getInput();
        auto& outPop = iteration.getOutput();

        Neon::index_3d dim(config.N, config.N, config.N);

        const auto& t = lattice.t_vect;
        const auto& c = lattice.c_vect;

        flag.forEachActiveCell([&](const Neon::index_3d& idx,
                                   const int&,
                                   CellType& flagVal) {
            flagVal.classification = CellType::undefined;
            flagVal.wallNghBitflag = 0;
            flagVal.classification = getBoundaryType(idx);

            bcTypeForDebugging.getReference(idx, 0) = static_cast<double>(flagVal.classification);
        });
        bcTypeForDebugging.ioToVtk("bcFlags", "cb", false);

        // Population initialization
        inPop.forEachActiveCell([&](const Neon::index_3d& idx,
                                    const int&            k,
                                    StorageFP&            val) {
            val = t.at(k);
            if (flag(idx, 0).classification == CellType::bounceBack) {
                val = 0;
            }
            if (flag(idx, 0).classification == CellType::pressure) {
                if (k == 0) {
                    flag.getReference(idx, 0).rho = rhoPrescribedOutlet;
                }
            }
            if (flag(idx, 0).classification == CellType::velocity) {
                if (k == 0) {
                    flag.getReference(idx, 0).rho = rhoPrescribedInlet;
                }
            }
        });

        inPop.forEachActiveCell([&](const Neon::index_3d& idx,
                                    const int&            k,
                                    StorageFP&            val) {

            if (flag(idx, 0).classification == CellType::pressure) {
                if (k == 0) {

                }
            }
            if (flag(idx, 0).classification == CellType::velocity) {
                if (k == 0) {
                    flag.getReference(idx, 0).rho = rhoPrescribedInlet;
                }
            }
        });

        outPop.forEachActiveCell([&](const Neon::index_3d& idx,
                                     const int&            k,
                                     StorageFP&            val) {
            val = t.at(k);
            if (flag(idx, 0).classification == CellType::bounceBack) {
                val = 0;
            }
            if (flag(idx, 0).classification == CellType::pressure) {
                if (k == 0) {
                    flag.getReference(idx, 0).rho = rhoPrescribedOutlet;
                }
            }
            if (flag(idx, 0).classification == CellType::velocity) {
                if (k == 0) {
                    flag.getReference(idx, 0).rho = rhoPrescribedInlet;
                }
            }
        });


        inPop.updateCompute(Neon::Backend::mainStreamIdx);
        outPop.updateCompute(Neon::Backend::mainStreamIdx);

        flag.updateCompute(Neon::Backend::mainStreamIdx);
        bk.syncAll();
        Neon::set::HuOptions hu(Neon::set::TransferMode::get,
                                false,
                                Neon::Backend::mainStreamIdx,
                                Neon::set::StencilSemantic::standard);

        flag.haloUpdate(hu);
        bk.syncAll();
        auto container = LbmContainers<Lattice, PopulationField, ComputeFP>::computeWallNghMask(flag, flag);
        container.run(Neon::Backend::mainStreamIdx);
        bk.syncAll();
    }

    metrics::recordProblemSetupMetrics(bk, report, start);

    // Reset the clock, to be used when a benchmark simulation is executed.
    tie(start, clock_iter) = metrics::restartClock(bk, true);

    int time_iter = 0;

    // The average energy, dependent on time, can be used to monitor convergence, or statistical
    // convergence, of the simulation.
    // Maximum number of time iterations depending on whether the simulation is in benchmark mode or production mode.
    // int max_time_iter = config.benchmark ? config.benchMaxIter : static_cast<int>(config.max_t / config.mLbmParameters.dt);
    int max_time_iter = config.benchMaxIter;

    for (time_iter = 0; time_iter < max_time_iter; ++time_iter) {
        if (!config.benchmark) {
            exportRhoAndU(time_iter);
        }

        if (config.benchmark && time_iter == config.benchIniIter) {
            std::cout << "Warm up completed (" << time_iter << " iterations ).\n"
                      << "Starting benchmark step ("
                      << config.benchMaxIter - config.benchIniIter << " iterations)."
                      << std::endl;
            tie(start, clock_iter) = metrics::restartClock(bk, false);
        }

        iteration.run();

        ++clock_iter;
    }
    std::cout << "Iterations completed" << std::endl;
    metrics::recordMetrics(bk, config, report, start, clock_iter);
}

template <typename Grid, typename StorageFP>
auto runFilterComputeType(Config& config, Report& report) -> void
{
    if (config.computeType == "double") {
        return runSpecialized<Grid, StorageFP, double>(config, report);
    }
    if (config.computeType == "float") {
        return runSpecialized<Grid, StorageFP, float>(config, report);
    }
    NEON_DEV_UNDER_CONSTRUCTION("");
}

template <typename Grid>
auto runFilterStoreType(Config& config,
                        Report& report)
    -> void
{
    if (config.storeType == "double") {
        return runFilterComputeType<Neon::domain::dGrid, double>(config, report);
    }
    if (config.storeType == "float") {
        return runFilterComputeType<Neon::domain::dGrid, float>(config, report);
    }
    NEON_DEV_UNDER_CONSTRUCTION("");
}
}  // namespace details

auto runTwoPop(Config& config,
               Report& report) -> void
{
    if (config.gridType == "dGrid") {
        return details::runFilterStoreType<Neon::domain::dGrid>(config, report);
    }
    if (config.gridType == "eGrid") {
        NEON_DEV_UNDER_CONSTRUCTION("");
    }
    if (config.gridType == "bGrid") {
        NEON_DEV_UNDER_CONSTRUCTION("");
    }
}
}  // namespace CavityTwoPop
