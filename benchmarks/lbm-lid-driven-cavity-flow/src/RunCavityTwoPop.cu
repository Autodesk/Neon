#include "hip/hip_runtime.h"
#include "Config.h"
#include "D3Q19.h"
#include "Neon/domain/bGrid.h"
#include "Neon/domain/dGrid.h"
#include "Neon/domain/details/dGridSoA/dGridSoA.h"
#include "Neon/domain/eGrid.h"

#include "CellType.h"
#include "LbmSkeleton.h"
#include "Metrics.h"
#include "Repoert.h"

namespace CavityTwoPop {

int backendWasReported = false;

namespace details {
template <typename Grid,
          typename Storage_,
          typename Compute_>
auto run(Config& config,
         Report& report) -> void
{
    using Storage = Storage_;
    using Compute = Compute_;
    using Precision = Precision<Storage, Compute>;
    using Lattice = D3Q19<Precision>;
    using PopulationField = typename Grid::template Field<Storage, Lattice::Q>;

    using PopField = typename Grid::template Field<typename Precision::Storage, Lattice::Q>;
    using CellTypeField = typename Grid::template Field<CellType, 1>;

    using Idx = typename PopField::Idx;
    using RhoField = typename Grid::template Field<typename Precision::Storage, 1>;
    using UField = typename Grid::template Field<typename Precision::Storage, 3>;

    using Skeleton = LbmSkeleton<Precision, Lattice, Grid>;
    using ContainerFactory = ContainerFactory<Precision, Lattice, Grid>;

    Neon::Backend bk = [&] {
        if (config.deviceType == "cpu") {
            Neon::Backend bk(config.devices, Neon::Runtime::openmp);
            return bk;
        }
        if (config.deviceType == "gpu") {
            Neon::Backend bk(config.devices, Neon::Runtime::stream);
            return bk;
        }
        Neon::NeonException exce("run");
        exce << config.deviceType << " is not a supported option as device type";
        NEON_THROW(exce);
    }();

    if (!backendWasReported) {
        metrics::recordBackend(bk, report);
        backendWasReported = true;
    }

    Neon::double_3d ulid(1., 0., 0.);
    // Neon Grid and Fields initialization
    auto [start, clock_iter] = metrics::restartClock(bk, true);
    Grid grid(
        bk, {config.N, config.N, config.N},
        [](const Neon::index_3d&) { return true; },
        Lattice::template getDirectionAsVector<Lattice::MemoryMapping>(),
        0.0, 1.0,
        config.spaceCurve);

    PopulationField pop0 = grid.template newField<Storage, Lattice::Q>("Population", Lattice::Q, Storage(0.0));
    PopulationField pop1 = grid.template newField<Storage, Lattice::Q>("Population", Lattice::Q, Storage(0.0));

    typename Grid::template Field<Storage, 1> rho;
    typename Grid::template Field<Storage, 3> u;

    if (!config.benchmark) {
        std::cout << "Allocating rho and u" << std::endl;
        rho = grid.template newField<Storage, 1>("rho", 1, Storage(0.0));
        u = grid.template newField<Storage, 3>("u", 3, Storage(0.0));
    }


    CellType defaultCelltype;
    auto     flag = grid.template newField<CellType, 1>("Material", 1, defaultCelltype);
    auto     lbmParameters = config.getLbmParameters<Compute>();

    Skeleton iteration(config.stencilSemantic,
                       config.occ,
                       config.transferMode,
                       pop0,
                       pop1,
                       flag,
                       lbmParameters.omega);

    auto exportRhoAndU = [&bk, &rho, &u, &iteration, &flag, &grid, &ulid](int iterationId) {
        if ((iterationId) % 100 == 0) {
            auto& f = iteration.getInput();
            {
                bk.syncAll();
                f.newHaloUpdate(Neon::set::StencilSemantic::standard,
                                Neon::set::TransferMode::get,
                                Neon::Execution::device)
                    .run(Neon::Backend::mainStreamIdx);
                bk.syncAll();
            }

            auto container = ContainerFactory::computeRhoAndU(f, flag, rho, u);
            container.run(Neon::Backend::mainStreamIdx);
            u.updateHostData(Neon::Backend::mainStreamIdx);
            rho.updateHostData(Neon::Backend::mainStreamIdx);
            // iteration.getInput().updateHostData(Neon::Backend::mainStreamIdx);

            bk.syncAll();
            size_t      numDigits = 5;
            std::string iterIdStr = std::to_string(iterationId);
            iterIdStr = std::string(numDigits - std::min(numDigits, iterIdStr.length()), '0') + iterIdStr;

            u.ioToVtk("u_" + iterIdStr, "u", false);
            rho.ioToVtk("rho_" + iterIdStr, "rho", false);
            // iteration.getInput().ioToVtk("pop_" + iterIdStr, "u", false);
            // flag.ioToVtk("flag_" + iterIdStr, "u", false);

            std::vector<std::pair<double, double>> xPosVal;
            std::vector<std::pair<double, double>> yPosVal;

            const double scale = 1.0 / ulid.v[0];

            const Neon::index_3d grid_dim = grid.getDimension();
            u.forEachActiveCell([&](const Neon::index_3d& id, const int& card, auto& val) {
                if (id.x == grid_dim.x / 2 && id.z == grid_dim.z / 2) {
                    if (card == 0) {
                        yPosVal.push_back({static_cast<double>(id.v[1]) / static_cast<double>(grid_dim.y), val * scale});
                    }
                }

                if (id.y == grid_dim.y / 2 && id.z == grid_dim.z / 2) {
                    if (card == 1) {
                        xPosVal.push_back({static_cast<double>(id.v[0]) / static_cast<double>(grid_dim.x), val * scale});
                    }
                }
            },
                                Neon::computeMode_t::seq);

            // sort the position so the linear interpolation works
            std::sort(xPosVal.begin(), xPosVal.end(), [=](std::pair<double, double>& a, std::pair<double, double>& b) {
                return a.first < b.first;
            });

            std::sort(yPosVal.begin(), yPosVal.end(), [=](std::pair<double, double>& a, std::pair<double, double>& b) {
                return a.first < b.first;
            });

            auto writeToFile = [](const std::vector<std::pair<double, double>>& posVal, std::string filename) {
                std::ofstream file;
                file.open(filename);
                for (auto v : posVal) {
                    file << v.first << " " << v.second << "\n";
                }
                file.close();
            };
            writeToFile(yPosVal, "NeonUniformLBM_" + iterIdStr + "_Y.dat");
            writeToFile(xPosVal, "NeonUniformLBM_" + iterIdStr + "_X.dat");
        }
    };


    metrics::recordGridInitMetrics(bk, report, start);
    tie(start, clock_iter) = metrics::restartClock(bk, true);

    // Problem Setup
    // 1. init all lattice to equilibrium
    {
        auto& inPop = iteration.getInput();
        auto& outPop = iteration.getOutput();

        Neon::index_3d dim(config.N, config.N, config.N);

        //        const auto& t = Lattice::Memory::t;
        //        const auto& c = Lattice::Memory::stencil;

        ContainerFactory::problemSetup(inPop,
                                       outPop,
                                       flag,
                                       ulid,
                                       config.ulb)
            .run(Neon::Backend::mainStreamIdx);


        inPop.updateHostData(Neon::Backend::mainStreamIdx);
        outPop.updateHostData(Neon::Backend::mainStreamIdx);
        flag.updateHostData(Neon::Backend::mainStreamIdx);
        {
            bk.syncAll();
            flag.newHaloUpdate(Neon::set::StencilSemantic::standard /*semantic*/,
                               Neon::set::TransferMode::get /*transferMode*/,
                               Neon::Execution::device /*execution*/)
                .run(Neon::Backend::mainStreamIdx);
            bk.syncAll();
        }

        auto container = ContainerFactory::computeWallNghMask(flag, flag);
        container.run(Neon::Backend::mainStreamIdx);
        bk.syncAll();
    }

    metrics::recordProblemSetupMetrics(bk, report, start);

    // Reset the clock, to be used when a benchmark simulation is executed.
    tie(start, clock_iter) = metrics::restartClock(bk, true);

    int time_iter = 0;

    // The average energy, dependent on time, can be used to monitor convergence, or statistical
    // convergence, of the simulation.
    // Maximum number of time iterations depending on whether the simulation is in benchmark mode or production mode.
    // int max_time_iter = config.benchmark ? config.benchMaxIter : static_cast<int>(config.max_t / config.mLbmParameters.dt);
    int max_time_iter = config.benchMaxIter;

    for (time_iter = 0; time_iter < max_time_iter; ++time_iter) {
        if (!config.benchmark) {
            exportRhoAndU(time_iter);
        }

        if (config.benchmark && time_iter == config.benchIniIter) {
            std::cout << "Warm up completed (" << time_iter << " iterations ).\n"
                      << "Starting benchmark step ("
                      << config.benchMaxIter - config.benchIniIter << " iterations)."
                      << std::endl;
            tie(start, clock_iter) = metrics::restartClock(bk, false);
        }

        iteration.run();

        ++clock_iter;
    }
    std::cout << "Iterations completed" << std::endl;
    metrics::recordMetrics(bk, config, report, start, clock_iter);
}

template <typename Grid, typename Storage>
auto runFilterComputeType(Config& config, Report& report) -> void
{
    if (config.computeType == "double") {
        return run<Grid, Storage, double>(config, report);
    }
    if (config.computeType == "float") {
        return run<Grid, Storage, float>(config, report);
    }
    NEON_DEV_UNDER_CONSTRUCTION("");
}

template <typename Grid>
auto runFilterStoreType(Config& config,
                        Report& report)
    -> void
{
    if (config.storeType == "double") {
        return runFilterComputeType<Grid, double>(config, report);
    }
    if (config.storeType == "float") {
        return runFilterComputeType<Grid, float>(config, report);
    }
}
}  // namespace details

#ifdef NEON_BENCHMARK_DESIGN_OF_EXPERIMENTS
constexpr bool skipTest = false;
#else
constexpr bool skipTest = false;
#endif

auto run(Config& config,
         Report& report) -> void
{
    if (config.gridType == "dGrid") {
        return details::runFilterStoreType<Neon::dGrid>(config, report);
    }
    if (config.gridType == "eGrid") {
        if constexpr (!skipTest) {
            return details::runFilterStoreType<Neon::eGrid>(config, report);
        } else {
            NEON_THROW_UNSUPPORTED_OPERATION("This option was disables. PLease define NEON_BENCHMARK_DESIGN_OF_EXPERIMENTS to enable it.")
        }
    }
    if (config.gridType == "bGrid") {
        return details::runFilterStoreType<Neon::bGrid>(config, report);
    }
    if (config.gridType == "bGrid_4_4_4") {
        if constexpr (!skipTest) {
            using Sblock = Neon::domain::details::bGrid::StaticBlock<4, 4, 4>;
            using Grid = Neon::domain::details::bGrid::bGrid<Sblock>;
            return details::runFilterStoreType<Grid>(config, report);
        } else {
            NEON_THROW_UNSUPPORTED_OPERATION("This option was disables. PLease define NEON_BENCHMARK_DESIGN_OF_EXPERIMENTS to enable it.")
        }
    }
    if (config.gridType == "bGrid_2_2_2") {
        if constexpr (!skipTest) {
            using Sblock = Neon::domain::details::bGrid::StaticBlock<2, 2, 2>;
            using Grid = Neon::domain::details::bGrid::bGrid<Sblock>;
            return details::runFilterStoreType<Grid>(config, report);
        } else {
            NEON_THROW_UNSUPPORTED_OPERATION("This option was disables. PLease define NEON_BENCHMARK_DESIGN_OF_EXPERIMENTS to enable it.")
        }
    }
    if (config.gridType == "bGrid_32_8_4") {
        if constexpr (!skipTest) {
            using Sblock = Neon::domain::details::bGrid::StaticBlock<32, 8, 4>;
            using Grid = Neon::domain::details::bGrid::bGrid<Sblock>;
            return details::runFilterStoreType<Grid>(config, report);
        } else {
            NEON_THROW_UNSUPPORTED_OPERATION("This option was disables. PLease define NEON_BENCHMARK_DESIGN_OF_EXPERIMENTS to enable it.")
        }
    }
    if (config.gridType == "bGrid_32_8_4") {
        if constexpr (!skipTest) {
            using Sblock = Neon::domain::details::bGrid::StaticBlock<32, 8, 4>;
            using Grid = Neon::domain::details::bGrid::bGrid<Sblock>;
            return details::runFilterStoreType<Grid>(config, report);
        } else {
            NEON_THROW_UNSUPPORTED_OPERATION("This option was disables. PLease define NEON_BENCHMARK_DESIGN_OF_EXPERIMENTS to enable it.")
        }
    }
    if (config.gridType == "bGrid_32_2_8") {
        if constexpr (!skipTest) {
            using Sblock = Neon::domain::details::bGrid::StaticBlock<32, 2, 8>;
            using Grid = Neon::domain::details::bGrid::bGrid<Sblock>;
            return details::runFilterStoreType<Grid>(config, report);
        } else {
            NEON_THROW_UNSUPPORTED_OPERATION("This option was disables. PLease define NEON_BENCHMARK_DESIGN_OF_EXPERIMENTS to enable it.")
        }
    }
    if (config.gridType == "bGrid_32_8_2") {
        if constexpr (!skipTest) {
            using Sblock = Neon::domain::details::bGrid::StaticBlock<32, 8, 2>;
            using Grid = Neon::domain::details::bGrid::bGrid<Sblock>;
            return details::runFilterStoreType<Grid>(config, report);
        } else {
            NEON_THROW_UNSUPPORTED_OPERATION("This option was disables. PLease define NEON_BENCHMARK_DESIGN_OF_EXPERIMENTS to enable it.")
        }
    }
    if (config.gridType == "dGridSoA") {
        if constexpr (!skipTest) {
            return details::runFilterStoreType<Neon::domain::details::dGridSoA::dGridSoA>(config, report);
        } else {
            NEON_THROW_UNSUPPORTED_OPERATION("This option was disables. PLease define NEON_BENCHMARK_DESIGN_OF_EXPERIMENTS to enable it.")
        }
    }
}
}  // namespace CavityTwoPop
