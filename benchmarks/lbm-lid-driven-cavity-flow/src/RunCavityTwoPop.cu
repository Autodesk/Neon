#include "hip/hip_runtime.h"
#include "Config.h"
#include "D3Q19.h"
#include "Neon/domain/bGrid.h"
#include "Neon/domain/dGrid.h"
#include "Neon/domain/details/dGridSoA/dGridSoA.h"
#include "Neon/domain/eGrid.h"

#include "CellType.h"
#include "LbmIteration.h"
#include "Metrics.h"
#include "Repoert.h"

namespace CavityTwoPop {

int backendWasReported = false;

namespace details {
template <typename Grid,
          typename StorageFP,
          typename ComputeFP>
auto run(Config& config,
         Report& report) -> void
{
    using Lattice = D3Q19<StorageFP, ComputeFP>;
    using PopulationField = typename Grid::template Field<StorageFP, Lattice::Q>;


    Neon::Backend bk = [&] {
        if (config.deviceType == "cpu") {
            Neon::Backend bk(config.devices, Neon::Runtime::openmp);
            return bk;
        }
        if (config.deviceType == "gpu") {
            Neon::Backend bk(config.devices, Neon::Runtime::stream);
            return bk;
        }
        Neon::NeonException exce("run");
        exce << config.deviceType << " is not a supported option as device type";
        NEON_THROW(exce);
    }();

    if (!backendWasReported) {
        metrics::recordBackend(bk, report);
        backendWasReported = true;
    }

    Neon::double_3d ulid(1., 0., 0.);
    Lattice         lattice(bk);

    // Neon Grid and Fields initialization
    auto [start, clock_iter] = metrics::restartClock(bk, true);
    Grid grid(
        bk, {config.N, config.N, config.N},
        [](const Neon::index_3d&) { return true; },
        lattice.getDirectionAsVector());

    PopulationField pop0 = grid.template newField<StorageFP, Lattice::Q>("Population", Lattice::Q, StorageFP(0.0));
    PopulationField pop1 = grid.template newField<StorageFP, Lattice::Q>("Population", Lattice::Q, StorageFP(0.0));

    typename Grid::template Field<StorageFP, 1> rho;
    typename Grid::template Field<StorageFP, 3> u;

    if (!config.benchmark) {
        std::cout << "Allocating rho and u" << std::endl;
        rho = grid.template newField<StorageFP, 1>("rho", 1, StorageFP(0.0));
        u = grid.template newField<StorageFP, 3>("u", 3, StorageFP(0.0));
    }


    CellType defaultCelltype;
    auto     flag = grid.template newField<CellType, 1>("Material", 1, defaultCelltype);
    auto     lbmParameters = config.getLbmParameters<ComputeFP>();

    LbmIterationD3Q19<PopulationField, ComputeFP>
        iteration(config.stencilSemantic,
                  config.occ,
                  config.transferMode,
                  pop0,
                  pop1,
                  flag,
                  lbmParameters.omega);

    auto exportRhoAndU = [&bk, &rho, &u, &iteration, &flag, &grid, &ulid](int iterationId) {
        if ((iterationId) % 100 == 0) {
            auto& f = iteration.getInput();
            {
                bk.syncAll();
                f.newHaloUpdate(Neon::set::StencilSemantic::standard,
                                Neon::set::TransferMode::get,
                                Neon::Execution::device)
                    .run(Neon::Backend::mainStreamIdx);
                bk.syncAll();
            }

            auto container = LbmContainers<Lattice, PopulationField, ComputeFP>::computeRhoAndU(f, flag, rho, u);
            container.run(Neon::Backend::mainStreamIdx);
            u.updateHostData(Neon::Backend::mainStreamIdx);
            rho.updateHostData(Neon::Backend::mainStreamIdx);
            // iteration.getInput().updateHostData(Neon::Backend::mainStreamIdx);

            bk.syncAll();
            size_t      numDigits = 5;
            std::string iterIdStr = std::to_string(iterationId);
            iterIdStr = std::string(numDigits - std::min(numDigits, iterIdStr.length()), '0') + iterIdStr;

            u.ioToVtk("u_" + iterIdStr, "u", false);
            rho.ioToVtk("rho_" + iterIdStr, "rho", false);
            // iteration.getInput().ioToVtk("pop_" + iterIdStr, "u", false);
            // flag.ioToVtk("flag_" + iterIdStr, "u", false);

            std::vector<std::pair<double, double>> xPosVal;
            std::vector<std::pair<double, double>> yPosVal;

            const double scale = 1.0 / ulid.v[0];

            const Neon::index_3d grid_dim = grid.getDimension();
            u.forEachActiveCell([&](const Neon::index_3d& id, const int& card, auto& val) {
                if (id.x == grid_dim.x / 2 && id.z == grid_dim.z / 2) {
                    if (card == 0) {
                        yPosVal.push_back({static_cast<double>(id.v[1]) / static_cast<double>(grid_dim.y), val * scale});
                    }
                }

                if (id.y == grid_dim.y / 2 && id.z == grid_dim.z / 2) {
                    if (card == 1) {
                        xPosVal.push_back({static_cast<double>(id.v[0]) / static_cast<double>(grid_dim.x), val * scale});
                    }
                }
            },
                                Neon::computeMode_t::seq);

            // sort the position so the linear interpolation works
            std::sort(xPosVal.begin(), xPosVal.end(), [=](std::pair<double, double>& a, std::pair<double, double>& b) {
                return a.first < b.first;
            });

            std::sort(yPosVal.begin(), yPosVal.end(), [=](std::pair<double, double>& a, std::pair<double, double>& b) {
                return a.first < b.first;
            });

            auto writeToFile = [](const std::vector<std::pair<double, double>>& posVal, std::string filename) {
                std::ofstream file;
                file.open(filename);
                for (auto v : posVal) {
                    file << v.first << " " << v.second << "\n";
                }
                file.close();
            };
            writeToFile(yPosVal, "NeonUniformLBM_" + iterIdStr + "_Y.dat");
            writeToFile(xPosVal, "NeonUniformLBM_" + iterIdStr + "_X.dat");
        }
    };


    metrics::recordGridInitMetrics(bk, report, start);
    tie(start, clock_iter) = metrics::restartClock(bk, true);

    // Problem Setup
    // 1. init all lattice to equilibrium
    {
        auto& inPop = iteration.getInput();
        auto& outPop = iteration.getOutput();

        Neon::index_3d dim(config.N, config.N, config.N);

        const auto& t = lattice.t_vect;
        const auto& c = lattice.c_vect;

        inPop.forEachActiveCell([&c, &t, &dim, &flag, &ulid, &config](const Neon::index_3d& idx,
                                                                      const int&            k,
                                                                      StorageFP&            val) {
            val = t.at(k);

            if (idx.x == 0 || idx.x == dim.x - 1 ||
                idx.y == 0 || idx.y == dim.y - 1 ||
                idx.z == 0 || idx.z == dim.z - 1) {

                if (idx.y == dim.y - 1) {
                    val = -6. * t.at(k) * config.ulb *
                          (c.at(k).v[0] * ulid.v[0] +
                           c.at(k).v[1] * ulid.v[1] +
                           c.at(k).v[2] * ulid.v[2]);
                } else {
                    val = 0;
                }
            }
        });

        outPop.forEachActiveCell([&c, &t, &dim, &flag, &ulid, &config](const Neon::index_3d& idx,
                                                                       const int&            k,
                                                                       StorageFP&            val) {
            val = t.at(k);

            if (idx.x == 0 || idx.x == dim.x - 1 ||
                idx.y == 0 || idx.y == dim.y - 1 ||
                idx.z == 0 || idx.z == dim.z - 1) {

                if (idx.y == dim.y - 1) {
                    val = -6. * t.at(k) * config.ulb *
                          (c.at(k).v[0] * ulid.v[0] +
                           c.at(k).v[1] * ulid.v[1] +
                           c.at(k).v[2] * ulid.v[2]);
                } else {
                    val = 0;
                }
            }
        });

        flag.forEachActiveCell([&dim](const Neon::index_3d& idx,
                                      const int&,
                                      CellType& flagVal) {
            flagVal.classification = CellType::bulk;
            flagVal.wallNghBitflag = 0;

            if (idx.x == 0 || idx.x == dim.x - 1 ||
                idx.y == 0 || idx.y == dim.y - 1 ||
                idx.z == 0 || idx.z == dim.z - 1) {

                flagVal.classification = CellType::bounceBack;

                if (idx.y == dim.y - 1) {
                    flagVal.classification = CellType::movingWall;
                }
            }
        });

        inPop.updateDeviceData(Neon::Backend::mainStreamIdx);
        outPop.updateDeviceData(Neon::Backend::mainStreamIdx);

        flag.updateDeviceData(Neon::Backend::mainStreamIdx);
        {
            bk.syncAll();
            flag.newHaloUpdate(Neon::set::StencilSemantic::standard /*semantic*/,
                               Neon::set::TransferMode::get /*transferMode*/,
                               Neon::Execution::device /*execution*/)
                .run(Neon::Backend::mainStreamIdx);
            bk.syncAll();
        }

        auto container = LbmContainers<Lattice, PopulationField, ComputeFP>::computeWallNghMask(flag, flag);
        container.run(Neon::Backend::mainStreamIdx);
        bk.syncAll();
    }

    metrics::recordProblemSetupMetrics(bk, report, start);

    // Reset the clock, to be used when a benchmark simulation is executed.
    tie(start, clock_iter) = metrics::restartClock(bk, true);

    int time_iter = 0;

    // The average energy, dependent on time, can be used to monitor convergence, or statistical
    // convergence, of the simulation.
    // Maximum number of time iterations depending on whether the simulation is in benchmark mode or production mode.
    // int max_time_iter = config.benchmark ? config.benchMaxIter : static_cast<int>(config.max_t / config.mLbmParameters.dt);
    int max_time_iter = config.benchMaxIter;

    for (time_iter = 0; time_iter < max_time_iter; ++time_iter) {
        if (!config.benchmark) {
            exportRhoAndU(time_iter);
        }

        if (config.benchmark && time_iter == config.benchIniIter) {
            std::cout << "Warm up completed (" << time_iter << " iterations ).\n"
                      << "Starting benchmark step ("
                      << config.benchMaxIter - config.benchIniIter << " iterations)."
                      << std::endl;
            tie(start, clock_iter) = metrics::restartClock(bk, false);
        }

        iteration.run();

        ++clock_iter;
    }
    std::cout << "Iterations completed" << std::endl;
    metrics::recordMetrics(bk, config, report, start, clock_iter);
}

template <typename Grid, typename StorageFP>
auto runFilterComputeType(Config& config, Report& report) -> void
{
    if (config.computeType == "double") {
        return run<Grid, StorageFP, double>(config, report);
    }
    if (config.computeType == "float") {
        return run<Grid, StorageFP, float>(config, report);
    }
    NEON_DEV_UNDER_CONSTRUCTION("");
}

template <typename Grid>
auto runFilterStoreType(Config& config,
                        Report& report)
    -> void
{
    if (config.storeType == "double") {
        return runFilterComputeType<Grid, double>(config, report);
    }
    if (config.storeType == "float") {
        return runFilterComputeType<Grid, float>(config, report);
    }
}
}  // namespace details

auto run(Config& config,
         Report& report) -> void
{
    if (config.gridType == "dGrid") {
        return details::runFilterStoreType<Neon::dGrid>(config, report);
    }
    if (config.gridType == "eGrid") {
        return details::runFilterStoreType<Neon::eGrid>(config, report);
    }
    if (config.gridType == "bGrid") {
        return details::runFilterStoreType<Neon::bGrid>(config, report);
    }
    if (config.gridType == "bGrid_4_4_4") {
        using Sblock = Neon::domain::details::bGrid::StaticBlock<4, 4, 4>;
        using Grid = Neon::domain::details::bGrid::bGrid<Sblock>;
        return details::runFilterStoreType<Grid>(config, report);
    }
    if (config.gridType == "bGrid_32_8_4") {
        using Sblock = Neon::domain::details::bGrid::StaticBlock<32, 8, 4>;
        using Grid = Neon::domain::details::bGrid::bGrid<Sblock>;
        return details::runFilterStoreType<Grid>(config, report);
    }
    if (config.gridType == "bGrid_32_8_4") {
        using Sblock = Neon::domain::details::bGrid::StaticBlock<32, 4, 8>;
        using Grid = Neon::domain::details::bGrid::bGrid<Sblock>;
        return details::runFilterStoreType<Grid>(config, report);
    }
    if (config.gridType == "bGrid_32_2_8") {
        using Sblock = Neon::domain::details::bGrid::StaticBlock<32, 2, 8>;
        using Grid = Neon::domain::details::bGrid::bGrid<Sblock>;
        return details::runFilterStoreType<Grid>(config, report);
    }
    if (config.gridType == "bGrid_32_8_2") {
        using Sblock = Neon::domain::details::bGrid::StaticBlock<32, 8, 2>;
        using Grid = Neon::domain::details::bGrid::bGrid<Sblock>;
        return details::runFilterStoreType<Grid>(config, report);
    }
    if (config.gridType == "dGridSoA") {
        return details::runFilterStoreType<Neon::domain::details::dGridSoA::dGridSoA>(config, report);
    }
}
}  // namespace CavityTwoPop
