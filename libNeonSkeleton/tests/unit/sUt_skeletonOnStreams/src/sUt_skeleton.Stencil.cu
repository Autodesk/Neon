#include <hip/hip_runtime_api.h>

#include "Neon/core/types/chrono.h"

#include "Neon/set/Containter.h"

#include "Neon/domain/bGrid.h"
#include "Neon/domain/tools/Geometries.h"
#include "Neon/domain/tools/TestData.h"

#include "Neon/skeleton/Options.h"
#include "Neon/skeleton/Skeleton.h"

#include <cctype>
#include <string>

#include "gtest/gtest.h"
#include "sUt.runHelper.h"

using namespace Neon::domain::tool::testing;
static const std::string testFilePrefix("sUt_skeleton_MapStencilMap");

template <typename Field, typename T>
auto axpy(const T&     val,
          const Field& y,
          Field&       x,
          size_t       sharedMem = 0) -> Neon::set::Container
{
    return y.getGrid().newContainer(
        "AXPY",
        y.getGrid().getDefaultBlock(),
        sharedMem,
        [&](Neon::set::Loader& L) -> auto {
            auto& yLocal = L.load(y);
            auto& xLocal = L.load(x);
            return [=] NEON_CUDA_HOST_DEVICE(const typename Field::Idx& gidx) mutable {
                // Neon::sys::ShmemAllocator shmemAlloc;
                // yLocal.loadInSharedMemory(e, 1, shmemAlloc);

                // Neon::index_3d global = xLocal.mapToGlobal(e);

                for (int i = 0; i < yLocal.cardinality(); i++) {
                    xLocal(gidx, i) += val * yLocal(gidx, i);
                }
            };
        });
}

template <typename Field>
auto laplace(const Field& x, Field& y, size_t sharedMem = 0) -> Neon::set::Container
{
    return x.getGrid().newContainer(
        "Laplace",
        x.getGrid().getDefaultBlock(),
        sharedMem,
        [&](Neon::set::Loader& L) -> auto {
            auto& xLocal = L.load(x, Neon::Pattern::STENCIL);
            auto& yLocal = L.load(y);

            return [=] NEON_CUDA_HOST_DEVICE(const typename Field::Idx& gidx) mutable {
                // Neon::sys::ShmemAllocator shmemAlloc;
                // xLocal.loadInSharedMemory(cell, 1, shmemAlloc);


                using Type = typename Field::Type;
                for (int card = 0; card < xLocal.cardinality(); card++) {
                    typename Field::Type res = 0;
                    int                  count = 0;

                    auto checkNeighbor = [&](Neon::domain::NghData<Type>& neighbor) {
                        if (neighbor.isValid()) {
                            res += neighbor.getData();
                            count++;
                        }
                    };

                    for (int8_t nghIdx = 0; nghIdx < 6; ++nghIdx) {
                        auto neighbor = xLocal.getNghData(gidx, nghIdx, card);
                        checkNeighbor(neighbor);
                    }

                    yLocal(gidx, card) = xLocal(gidx, card) - count * res;
                }
            };
        });
}


template <typename G, typename T, int C>
void SingleStencil(TestData<G, T, C>&      data,
                   Neon::skeleton::Occ     occ,
                   Neon::set::TransferMode transfer)
{
    using Type = typename TestData<G, T, C>::Type;

    auto occName = Neon::skeleton::OccUtils::toString(occ);
    occName[0] = toupper(occName[0]);
    const std::string appName(testFilePrefix + "_" + occName);

    Neon::skeleton::Skeleton skl(data.getBackend());
    Neon::skeleton::Options  opt(occ, transfer);

    const int nIterations = 5;

    const T val = 89;

    data.getBackend().syncAll();

    data.resetValuesToRandom(1, 50);


    {  // SKELETON
        auto& X = data.getField(FieldNames::X);
        auto& Y = data.getField(FieldNames::Y);

        std::vector<Neon::set::Container> ops;


        /*X.forEachActiveCell([&](const Neon::index_3d& idx,
                                const int&            cardinality,
                                T&) {


        });
        X.updateDeviceData(0);
        X.ioToVtk("X", "X");*/

        ops.push_back(laplace(X, Y, 0));
        ops.push_back(axpy(val, Y, X, 0));

        skl.sequence(ops, appName, opt);

        NEON_CUDA_CHECK_LAST_ERROR
        hipProfilerStart();
        for (int i = 0; i < nIterations; i++) {
            skl.run();
        }
        data.getBackend().syncAll();
        hipProfilerStop();
        NEON_CUDA_CHECK_LAST_ERROR

        // X.ioToVtk("X", "X");
        // Y.ioToVtk("Y", "Y");
    }

    {  // Golden data
        auto& X = data.getIODomain(FieldNames::X);
        auto& Y = data.getIODomain(FieldNames::Y);

        for (int i = 0; i < nIterations; i++) {
            data.laplace(X, Y);
            data.axpy(&val, Y, X);
        }
    }

    bool isOk = data.compare(FieldNames::X);
    isOk = isOk && data.compare(FieldNames::Y);

    ASSERT_TRUE(isOk);
}

template <typename G, typename T, int C>
void SingleStencilOCC(TestData<G, T, C>& data)
{
    SingleStencil<G, T, C>(data, Neon::skeleton::Occ::standard, Neon::set::TransferMode::get);
}

template <typename G, typename T, int C>
void SingleStencilExtendedOCC(TestData<G, T, C>& data)
{
    SingleStencil<G, T, C>(data, Neon::skeleton::Occ::extended, Neon::set::TransferMode::get);
}

template <typename G, typename T, int C>
void SingleStencilTwoWayExtendedOCC(TestData<G, T, C>& data)
{
    SingleStencil<G, T, C>(data, Neon::skeleton::Occ::twoWayExtended, Neon::set::TransferMode::get);
}

template <typename G, typename T, int C>
void SingleStencilNoOCC(TestData<G, T, C>& data)
{
    SingleStencil<G, T, C>(data, Neon::skeleton::Occ::none, Neon::set::TransferMode::get);
}

namespace {
int getNGpus()
{
    if (Neon::sys::globalSpace::gpuSysObjStorage.numDevs() > 0) {
        int maxGPUs = Neon::set::DevSet::maxSet().setCardinality();
        if (maxGPUs > 1) {
            return maxGPUs;
        } else {
            return 3;
        }
    } else {
        return 0;
    }
}
}  // namespace


TEST(SingleStencil_NoOCC, bGrid)
{
    int nGpus = 1;
    using Grid = Neon::bGrid;
    // using Grid = Neon::domain::eGrid;
    // using Grid = Neon::dGrid;
    using Type = int32_t;
    runAllTestConfiguration<Grid, Type, 0>("bGrid_t", SingleStencilNoOCC<Grid, Type, 0>, nGpus, 1);
}