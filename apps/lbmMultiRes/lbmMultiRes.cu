#include "hip/hip_runtime.h"
#include "Neon/Neon.h"
#include "Neon/domain/mGrid.h"
#include "Neon/skeleton/Skeleton.h"

template <unsigned int DIM, unsigned int Q>
Neon::domain::Stencil create_stencil();

template <>
Neon::domain::Stencil create_stencil<2, 9>()
{
    std::vector<Neon::index_3d> stencil;
    stencil.reserve(9);
    for (int x = -1; x <= 1; ++x) {
        for (int y = -1; y <= 1; ++y) {
            stencil.emplace_back(Neon::index_3d(x, y, 0));
        }
    }
    return Neon::domain::Stencil(stencil);
}

template <>
Neon::domain::Stencil create_stencil<3, 19>()
{
    // filterCenterOut = false;
    return Neon::domain::Stencil::s19_t(false);
}

NEON_CUDA_DEVICE_ONLY static constexpr char latticeVelocity2D[9][2] = {
    {0, 0},
    {0, -1},
    {0, 1},
    {-1, 0},
    {-1, -1},
    {-1, 1},
    {1, 0},
    {1, -1},
    {1, 1}};

NEON_CUDA_DEVICE_ONLY static constexpr char latticeVelocity3D[27][3] = {
    {0, 0, 0},
    {0, -1, 0},
    {0, 1, 0},
    {-1, 0, 0},
    {-1, -1, 0},
    {-1, 1, 0},
    {1, 0, 0},
    {1, -1, 0},
    {1, 1, 0},

    {0, 0, -1},
    {0, -1, -1},
    {0, 1, -1},
    {-1, 0, -1},
    {-1, -1, -1},
    {-1, 1, -1},
    {1, 0, -1},
    {1, -1, -1},
    {1, 1, -1},

    {0, 0, 1},
    {0, -1, 1},
    {0, 1, 1},
    {-1, 0, 1},
    {-1, -1, 1},
    {-1, 1, 1},
    {1, 0, 1},
    {1, -1, 1},
    {1, 1, 1}

};


template <int DIM>
NEON_CUDA_HOST_DEVICE Neon::int8_3d getDir(const int8_t q)
{
    if constexpr (DIM == 2) {
        return Neon::int8_3d(latticeVelocity2D[q][0], latticeVelocity2D[q][1], 0);
    }
    if constexpr (DIM == 3) {
        return Neon::int8_3d(latticeVelocity3D[q][0], latticeVelocity3D[q][1], latticeVelocity3D[q][2]);
    }
}

template <int DIM, int Q>
struct latticeWeight
{
    NEON_CUDA_HOST_DEVICE __inline__ constexpr latticeWeight()
        : t()
    {
        if constexpr (DIM == 2) {

            for (int i = 0; i < Q; ++i) {
                if (latticeVelocity2D[i][0] * latticeVelocity2D[i][0] +
                        latticeVelocity2D[i][1] * latticeVelocity2D[i][1] <
                    1.1f) {
                    t[i] = 1.0f / 9.0f;
                } else {
                    t[i] = 1.0f / 36.0f;
                }
            }
            t[0] = 4.0f / 9.0f;
        }

        if constexpr (DIM == 3) {
            for (int i = 0; i < Q; ++i) {
                if (latticeVelocity2D[i][0] * latticeVelocity2D[i][0] +
                        latticeVelocity2D[i][1] * latticeVelocity2D[i][1] +
                        latticeVelocity2D[i][2] * latticeVelocity2D[i][2] <
                    1.1f) {
                    t[i] = 2.0f / 36.0f;
                } else {
                    t[i] = 1.0f / 36.0f;
                }
            }
            t[0] = 1.0f / 3.0f;
        }
    }
    float t[Q];
};


template <typename Field>
inline void exportVTI(const int t, Field& field)
{
    printf("\n Exporting Frame =%d", t);
    int                precision = 4;
    std::ostringstream oss;
    oss << std::setw(precision) << std::setfill('0') << t;
    std::string prefix = "lbm" + std::to_string(field.getCardinality()) + "D_";
    std::string fname = prefix + oss.str();
    field.ioToVtk(fname, "field");
}

template <typename T>
NEON_CUDA_HOST_DEVICE inline Neon::int8_3d unlceOffset(const T& cell, const Neon::int8_3d& q)
{
    //given a local index within a cell and a population direction (q)
    //find the uncle's (the parent neighbor) offset from which the desired population (q) should be read
    //this offset is wrt the cell containing the localID (i.e., the parent of localID)

    auto off = [](const int8_t i, const int8_t j) {
        //0, -1 --> -1
        //1, -1 --> 0
        //0, 0 --> 0
        //0, 1 --> 0
        //1, 1 --> 1
        const int8_t s = i + j;
        return (s <= 0) ? s : s - 1;
    };

    Neon::int8_3d offset(off(cell.x, q.x), off(cell.y, q.y), off(cell.z, q.z));
    return offset;
}


template <typename T>
NEON_CUDA_HOST_DEVICE T computeOmega(T omega0, int level, int num_levels)
{
    int ilevel = num_levels - level - 1;
    // scalbln(1.0, x) = 2^x
    return 2 * omega0 / (scalbln(1.0, ilevel + 1) + (1. - scalbln(1.0, ilevel)) * omega0);
}

template <typename T, int DIM, int Q>
NEON_CUDA_HOST_DEVICE Neon::Vec_3d<T> velocity(const T* fin,
                                               const T  rho)
{
    Neon::Vec_3d<T> vel(0, 0, 0);
    if constexpr (DIM == 2) {
        for (int i = 0; i < Q; ++i) {
            const T f = fin[i];
            for (int d = 0; d < DIM; ++d) {
                vel.v[d] += f * latticeVelocity2D[i][d];
            }
        }
    }

    if constexpr (DIM == 3) {
        for (int i = 0; i < Q; ++i) {
            const T f = fin[i];
            for (int d = 0; d < DIM; ++d) {
                vel.v[d] += f * latticeVelocity3D[i][d];
            }
        }
    }

    for (int d = 0; d < DIM; ++d) {
        vel.v[d] /= rho;
    }
    return vel;
}

template <typename T, int DIM, int Q>
Neon::set::Container collide(Neon::domain::mGrid&                 grid,
                             T                                    omega0,
                             int                                  level,
                             int                                  max_level,
                             const Neon::domain::mGrid::Field<T>& fin,
                             Neon::domain::mGrid::Field<T>&       fout)
{
    return grid.getContainer(
        "Collide" + std::to_string(level), level,
        [=](Neon::set::Loader& loader) {
            const auto& in = fin.load(loader, level, Neon::MultiResCompute::MAP);
            auto        out = fout.load(loader, level, Neon::MultiResCompute::MAP);
            const T     omega = computeOmega(omega0, level, max_level);

            return [=] NEON_CUDA_HOST_DEVICE(const typename Neon::domain::bGrid::Cell& cell) mutable {
                if (!in.hasChildren(cell)) {

                    constexpr auto t = latticeWeight<DIM, Q>();

                    //fin
                    T ins[Q];
                    for (int i = 0; i < Q; ++i) {
                        ins[i] = in(cell, i);
                    }

                    //density
                    T rho = 0;
                    for (int i = 0; i < Q; ++i) {
                        rho += ins[i];
                    }

                    //velocity
                    const Neon::Vec_3d<T> vel = velocity<T, DIM, Q>(ins, rho);


                    const T usqr = (3.0 / 2.0) * (vel.x * vel.x + vel.y * vel.y + vel.z * vel.z);
                    for (int i = 0; i < Q; ++i) {
                        T cu = 0;
                        for (int d = 0; d < DIM; ++d) {
                            cu += latticeVelocity2D[i][d] * vel.v[d];
                        }
                        //equilibrium
                        T feq = rho * t.t[i] * (1. + cu + 0.5 * cu * cu - usqr);

                        //collide
                        out(cell, i) = ins[i] - omega * (ins[i] - feq);
                    }
                }
            };
        });
}

template <typename T, int DIM, int Q>
Neon::set::Container stream(Neon::domain::mGrid&                 grid,
                            int                                  level,
                            const Neon::domain::mGrid::Field<T>& fpop_postcollision,
                            Neon::domain::mGrid::Field<T>&       fpop_poststreaming)
{
    //regular Streaming of the normal voxels at level L which are not interfaced with L+1 and L-1 levels.
    //This is "pull" stream

    return grid.getContainer(
        "Stream" + std::to_string(level), level,
        [=](Neon::set::Loader& loader) {
            const auto& fpost_col = fpop_postcollision.load(loader, level, Neon::MultiResCompute::STENCIL);
            auto        fpost_stm = fpop_poststreaming.load(loader, level, Neon::MultiResCompute::MAP);

            return [=] NEON_CUDA_HOST_DEVICE(const typename Neon::domain::bGrid::Cell& cell) mutable {
                //If this cell has children i.e., it is been refined, than we should not work on it
                //because this cell is only there to allow query and not to operate on
                if (!fpost_stm.hasChildren(cell)) {

                    for (int8_t q = 0; q < Q; ++q) {
                        const Neon::int8_3d dir = -getDir<DIM>(q);

                        //if the neighbor cell has children, then this 'cell' is interfacing with L-1 (fine) along q direction
                        if (!fpost_stm.hasChildren(cell, dir)) {
                            auto neighbor = fpost_col.nghVal(cell, dir, q, T(0));
                            if (neighbor.isValid) {
                                fpost_stm(cell, q) = neighbor.value;
                            }
                        }
                    }
                }
            };
        });
}

template <typename T, int DIM, int Q>
Neon::set::Container explosionPull(Neon::domain::mGrid&                 grid,
                                   int                                  level,
                                   const Neon::domain::mGrid::Field<T>& fpop_postcollision,
                                   Neon::domain::mGrid::Field<T>&       fpop_poststreaming)
{
    // Initiated by the fine level (hence "pull"), this function performs a coarse (level+1) to
    // fine (level) communication or "explosion" by simply distributing copies of coarse grid onto the fine grid.
    // In other words, this function updates the "halo" cells of the fine level by making copies of the coarse cell
    // values.


    return grid.getContainer(
        "explosionPull" + std::to_string(level), level,
        [=](Neon::set::Loader& loader) {
            const auto& fpost_col = fpop_postcollision.load(loader, level, Neon::MultiResCompute::STENCIL_UP);
            auto        fpost_stm = fpop_poststreaming.load(loader, level, Neon::MultiResCompute::MAP);

            return [=] NEON_CUDA_HOST_DEVICE(const typename Neon::domain::bGrid::Cell& cell) mutable {
                //If this cell has children i.e., it is been refined, that we should not work on it
                //because this cell is only there to allow query and not to operate on
                if (!fpost_stm.hasChildren(cell)) {
                    for (int8_t q = 0; q < Q; ++q) {

                        const Neon::int8_3d dir = -getDir<DIM>(q);

                        //if the neighbor cell has children, then this 'cell' is interfacing with L-1 (fine) along q direction
                        //we want to only work on cells that interface with L+1 (coarse) cell along q
                        if (!fpost_stm.hasChildren(cell, dir)) {

                            //try to query the cell along this direction (opposite of the population direction) as we do
                            //in 'normal' streaming
                            auto neighborCell = fpost_col.setNghCell(cell, dir);
                            if (!neighborCell.isActive()) {
                                //only if we can not do normal streaming, then we may have a coarser neighbor from which
                                //we can read this pop

                                //get the uncle direction/offset i.e., the neighbor of the cell's parent
                                //this direction/offset is wrt to the cell's parent
                                Neon::int8_3d uncleDir = unlceOffset(cell.mLocation, dir);

                                auto uncle = fpost_col.uncleVal(cell, uncleDir, q, T(0));
                                if (uncle.isValid) {
                                    fpost_stm(cell, q) = uncle.value;
                                }
                            }
                        }
                    }
                }
            };
        });
}


template <typename T, int DIM, int Q>
Neon::set::Container coalescencePull(Neon::domain::mGrid&           grid,
                                     int                            level,
                                     Neon::domain::mGrid::Field<T>& fpop_poststreaming)
{
    // Initiated by the coarse level (hence "pull"), this function simply read the missing population
    // across the interface between coarse<->fine boundary by reading the population prepare during the store()

    return grid.getContainer(
        "coalescencePull" + std::to_string(level), level,
        [=](Neon::set::Loader& loader) {
            auto fpost_stm = fpop_poststreaming.load(loader, level, Neon::MultiResCompute::STENCIL);

            return [=] NEON_CUDA_HOST_DEVICE(const typename Neon::domain::bGrid::Cell& cell) mutable {
                //If this cell has children i.e., it is been refined, than we should not work on it
                //because this cell is only there to allow query and not to operate on
                if (!fpost_stm.hasChildren(cell)) {

                    for (int q = 0; q < Q; ++q) {
                        const Neon::int8_3d dir = -getDir<DIM>(q);
                        //if we have a neighbor at the same level that has been refined, then cell is on
                        //the interface and this is where we should do the coalescence
                        if (fpost_stm.hasChildren(cell, dir)) {
                            auto neighbor = fpost_stm.nghVal(cell, dir, q, T(0));
                            if (neighbor.isValid) {
                                fpost_stm(cell, q) = neighbor.value;
                            }
                        }
                    }
                }
            };
        });
}


template <typename T, int DIM, int Q>
Neon::set::Container store(Neon::domain::mGrid&           grid,
                           int                            level,
                           Neon::domain::mGrid::Field<T>& fpop_poststreaming)
{
    //Initiated by the coarse level (level), this function prepares and stores the fine (level - 1)
    // information for further pulling initiated by the coarse (this) level invoked by coalescence_pull
    //
    //Where a coarse cell stores its information? at itself i.e., pull
    //Where a coarse cell reads the needed info? from its children and neighbor cell's children (level -1)
    //This function only operates on a coarse cell that has children.
    //For such cell, we check its neighbor cells at the same level. If any of these neighbor has NO
    //children, then we need to prepare something for them to be read during coalescence. What
    //we prepare is some sort of averaged the data from the children (the cell's children and/or
    //its neighbor's children)

    return grid.getContainer(
        "store" + std::to_string(level), level,
        [=](Neon::set::Loader& loader) {
            auto fpost_stm = fpop_poststreaming.load(loader, level, Neon::MultiResCompute::STENCIL_DOWN);

            return [=] NEON_CUDA_HOST_DEVICE(const typename Neon::domain::bGrid::Cell& cell) mutable {
                //if the cell is refined, we might need to store something in it for its neighbor
                if (fpost_stm.hasChildren(cell)) {

                    const int refFactor = fpost_stm.getRefFactor(level);

                    bool should_accumelate =  ((int(fpost_stm(cell, 0)) % refFactor) != 0);

                    fpost_stm(cell, 0) += 1;


                    //for each direction aka for each neighbor
                    //we skip the center here
                    for (int8_t q = 1; q < Q; ++q) {
                        const Neon::int8_3d q_dir = getDir<DIM>(q);

                        //check if the neighbor in this direction has children
                        if (!fpost_stm.hasChildren(cell, q_dir)) {
                            //now, we know that there is actually something we need to store for this neighbor
                            //in cell along q (q_dir) direction
                            int num = 0;
                            T   sum = 0;


                            //for every neighbor cell including the center cell (i.e., cell)
                            for (int8_t p = 0; p < Q; ++p) {
                                const Neon::int8_3d p_dir = getDir<DIM>(p);

                                //relative direction of q w.r.t p
                                //i.e., in which direction we should move starting from p to land on q
                                const Neon::int8_3d r_dir = q_dir - p_dir;

                                //if this neighbor is refined
                                if (fpost_stm.hasChildren(cell, p_dir)) {

                                    //for each children of p
                                    for (int8_t i = 0; i < refFactor; ++i) {
                                        for (int8_t j = 0; j < refFactor; ++j) {
                                            for (int8_t k = 0; k < refFactor; ++k) {
                                                const Neon::int8_3d c(i, j, k);

                                                //cq is coarse neighbor (i.e., uncle) that we need to go in order to read q
                                                //for c (this is what we do for explosion but here we do this just for the check)
                                                const Neon::int8_3d cq = unlceOffset(c, q_dir);
                                                if (cq == r_dir) {
                                                    num++;
                                                    sum += fpost_stm.childVal(cell, c, q, 0).value;
                                                }
                                            }
                                        }
                                    }
                                }
                            }

                            if (should_accumelate) {
                                fpost_stm(cell, q) += sum / static_cast<T>(num * refFactor);
                            } else {
                                fpost_stm(cell, q) = sum / static_cast<T>(num * refFactor);
                            }
                        }
                    }
                }
            };
        });
}


template <typename T, int DIM, int Q>
void stream(Neon::domain::mGrid&                 grid,
            int                                  level,
            const int                            max_level,
            const Neon::domain::mGrid::Field<T>& fpop_postcollision,
            Neon::domain::mGrid::Field<T>&       fpop_poststreaming,
            std::vector<Neon::set::Container>&   containers)
{
    containers.push_back(stream<T, DIM, Q>(grid, level, fpop_postcollision, fpop_poststreaming));

    /*
    * Streaming for interface voxels that have
    *  (i) coarser or (ii) finer neighbors at level+1 and level-1 and hence require
    *  (i) "explosion" or (ii) coalescence
    */
    if (level != max_level - 1) {
        /* Explosion: pull missing populations from coarser neighbors by copying coarse (level+1) to fine (level) 
        * neighbors, initiated by the fine level ("Pull").
        */
        containers.push_back(explosionPull<T, DIM, Q>(grid, level, fpop_postcollision, fpop_poststreaming));
    }

    if (level != 0) {
        /* Coalescence: pull missing populations from finer neighbors by "smart" averaging fine (level-1) 
        * to coarse (level) communication, initiated by the coarse level ("Pull").
        */
        containers.push_back(coalescencePull<T, DIM, Q>(grid, level, fpop_poststreaming));
    }
}

template <typename T, int DIM, int Q>
void nonUniformTimestepRecursive(Neon::domain::mGrid&               grid,
                                 const T                            omega0,
                                 const int                          level,
                                 const int                          max_level,
                                 Neon::domain::mGrid::Field<T>&     fin,
                                 Neon::domain::mGrid::Field<T>&     fout,
                                 std::vector<Neon::set::Container>& containers)
{
    // 1) collision for all voxels at level L=level
    containers.push_back(collide<T, DIM, Q>(grid, omega0, level, max_level, fin, fout));

    // 2) Storing fine (level - 1) data for later "coalescence" pulled by the coarse (level)
    if (level != max_level) {
        store<T, DIM, Q>(grid, level + 1, fout);
    }


    // 3) recurse down
    if (level != 0) {
        nonUniformTimestepRecursive<T, DIM, Q>(grid, omega0, level - 1, max_level, fin, fout, containers);
    }

    // 4) Streaming step that also performs the necessary "explosion" and "coalescence" steps.
    stream<T, DIM, Q>(grid, level, max_level, fout, fin, containers);

    // 5) stop
    if (level == max_level - 1) {
        return;
    }

    // 6) collision for all voxels at level L = level
    containers.push_back(collide<T, DIM, Q>(grid, omega0, level, max_level, fin, fout));

    // 7) Storing fine(level) data for later "coalescence" pulled by the coarse(level)
    if (level != max_level) {
        store<T, DIM, Q>(grid, level + 1, fout);
    }

    // 8) recurse down
    if (level != 0) {
        nonUniformTimestepRecursive<T, DIM, Q>(grid, omega0, level - 1, max_level, fin, fout, containers);
    }

    // 9) Streaming step
    stream<T, DIM, Q>(grid, level, max_level, fout, fin, containers);
}


int main(int argc, char** argv)
{
    Neon::init();

    if (Neon::sys::globalSpace::gpuSysObjStorage.numDevs() > 0) {
        using T = double;

        //Neon grid
        auto             runtime = Neon::Runtime::openmp;
        std::vector<int> gpu_ids{0};
        Neon::Backend    backend(gpu_ids, runtime);

        constexpr int DIM = 2;
        constexpr int Q = (DIM == 2) ? 9 : 19;

        const int dim_x = 12;
        const int dim_y = 12;
        const int dim_z = (DIM < 3) ? 4 : 4;

        const Neon::index_3d grid_dim(dim_x, dim_y, dim_z);

        const Neon::domain::mGridDescriptor descriptor({1, 1, 1});


        Neon::domain::mGrid grid(
            backend, grid_dim,
            {[&](const Neon::index_3d id) -> bool {
                 return id.x > 7;
             },
             [&](const Neon::index_3d& id) -> bool {
                 return id.x > 3;
             },
             [&](const Neon::index_3d&) -> bool {
                 return true;
             }},
            create_stencil<DIM, Q>(), descriptor);

        //grid.topologyToVTK("lbm.vtk", false);


        //LBM problem
        const int max_iter = 300;
        const T   ulb = 0.01;
        const T   Re = 20;
        const T   clength = grid_dim.x;
        const T   visclb = ulb * clength / Re;
        const T   smagorinskyConstant = 0.02;
        const T   omega = 1.0 / (3. * visclb + 0.5);

        auto fin = grid.newField<T>("fin", Q, 0);
        auto fout = grid.newField<T>("fout", Q, 0);

        //TODO init fin and fout
        for (int l = 0; l < descriptor.getDepth(); ++l) {
            fin.forEachActiveCell(
                l,
                [&](const Neon::int32_3d, const int, T& val) {
                    val = 0;
                });
            fout.forEachActiveCell(
                l,
                [&](const Neon::int32_3d, const int, T& val) {
                    val = 0;
                });
        }

        fin.updateCompute();
        fout.updateCompute();

        std::vector<Neon::set::Container> containers;

        nonUniformTimestepRecursive<T, DIM, Q>(grid,
                                               omega,
                                               descriptor.getDepth() - 1,
                                               descriptor.getDepth(),
                                               fin, fout, containers);

        Neon::skeleton::Skeleton skl(grid.getBackend());
        skl.sequence(containers, "MultiResLBM");
        //skl.ioToDot("MultiRes");

        skl.run();

        grid.getBackend().syncAll();
        fin.updateIO();
        fout.updateIO();
    }
}