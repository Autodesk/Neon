#include "hip/hip_runtime.h"
#include "Neon/Neon.h"
#include "Neon/domain/mGrid.h"
#include "Neon/skeleton/Skeleton.h"

template <unsigned int DIM, unsigned int Q>
Neon::domain::Stencil create_stencil();

template <>
Neon::domain::Stencil create_stencil<2, 9>()
{
    std::vector<Neon::index_3d> stencil;
    stencil.reserve(9);
    for (int x = -1; x <= 1; ++x) {
        for (int y = -1; y <= 1; ++y) {
            stencil.emplace_back(Neon::index_3d(x, y, 0));
        }
    }
    return Neon::domain::Stencil(stencil);
}

template <>
Neon::domain::Stencil create_stencil<3, 19>()
{
    // filterCenterOut = false;
    return Neon::domain::Stencil::s19_t(false);
}

NEON_CUDA_DEVICE_ONLY static constexpr char latticeVelocity2D[9][2] = {
    {0, 0},
    {0, -1},
    {0, 1},
    {-1, 0},
    {-1, -1},
    {-1, 1},
    {1, 0},
    {1, -1},
    {1, 1}};

NEON_CUDA_DEVICE_ONLY static constexpr char latticeVelocity3D[27][3] = {
    {0, 0, 0},
    {0, -1, 0},
    {0, 1, 0},
    {-1, 0, 0},
    {-1, -1, 0},
    {-1, 1, 0},
    {1, 0, 0},
    {1, -1, 0},
    {1, 1, 0},

    {0, 0, -1},
    {0, -1, -1},
    {0, 1, -1},
    {-1, 0, -1},
    {-1, -1, -1},
    {-1, 1, -1},
    {1, 0, -1},
    {1, -1, -1},
    {1, 1, -1},

    {0, 0, 1},
    {0, -1, 1},
    {0, 1, 1},
    {-1, 0, 1},
    {-1, -1, 1},
    {-1, 1, 1},
    {1, 0, 1},
    {1, -1, 1},
    {1, 1, 1}

};

template <int DIM, int Q>
struct latticeWeight
{
    NEON_CUDA_HOST_DEVICE __inline__ constexpr latticeWeight()
        : t()
    {
        if constexpr (DIM == 2) {

            for (int i = 0; i < Q; ++i) {
                if (latticeVelocity2D[i][0] * latticeVelocity2D[i][0] +
                        latticeVelocity2D[i][1] * latticeVelocity2D[i][1] <
                    1.1f) {
                    t[i] = 1.0f / 9.0f;
                } else {
                    t[i] = 1.0f / 36.0f;
                }
            }
            t[0] = 4.0f / 9.0f;
        }

        if constexpr (DIM == 3) {
            for (int i = 0; i < Q; ++i) {
                if (latticeVelocity2D[i][0] * latticeVelocity2D[i][0] +
                        latticeVelocity2D[i][1] * latticeVelocity2D[i][1] +
                        latticeVelocity2D[i][2] * latticeVelocity2D[i][2] <
                    1.1f) {
                    t[i] = 2.0f / 36.0f;
                } else {
                    t[i] = 1.0f / 36.0f;
                }
            }
            t[0] = 1.0f / 3.0f;
        }
    }
    float t[Q];
};


template <typename Field>
inline void exportVTI(const int t, Field& field)
{
    printf("\n Exporting Frame =%d", t);
    int                precision = 4;
    std::ostringstream oss;
    oss << std::setw(precision) << std::setfill('0') << t;
    std::string prefix = "lbm" + std::to_string(field.getCardinality()) + "D_";
    std::string fname = prefix + oss.str();
    field.ioToVtk(fname, "field");
}


NEON_CUDA_HOST_DEVICE inline Neon::int8_3d explosionUnlceOffset(const Neon::domain::bGrid::Cell& cell, const Neon::int8_3d& q)
{
    //given a local index within a cell and a population direction (q)
    //find the uncle's (the parent neighbor) offset from which the desired population (q) should be read
    //this offset is wrt the cell containing the localID (i.e., the parent of localID)

    auto off = [](const int8_t i, const int8_t j) {
        //0, -1 --> -1
        //1, -1 --> 0
        //0, 1 --> 0
        //0, 0 --> 0
        //1, 1 --> 1
        const int8_t s = i + j;
        return (s <= 0) ? s : s - 1;
    };

    Neon::int8_3d offset(off(cell.mLocation.x, q.x), off(cell.mLocation.y, q.y), off(cell.mLocation.z, q.z));
    return offset;
}


template <typename T>
NEON_CUDA_HOST_DEVICE T computeOmega(T omega0, int level, int num_levels)
{
    int ilevel = num_levels - level - 1;
    // scalbln(1.0, x) = 2^x
    return 2 * omega0 / (scalbln(1.0, ilevel + 1) + (1. - scalbln(1.0, ilevel)) * omega0);
}

template <typename T, int DIM, int Q>
NEON_CUDA_HOST_DEVICE Neon::Vec_3d<T> velocity(const T* fin,
                                               const T  rho)
{
    Neon::Vec_3d<T> vel(0, 0, 0);
    if constexpr (DIM == 2) {
        for (int i = 0; i < Q; ++i) {
            const T f = fin[i];
            for (int d = 0; d < DIM; ++d) {
                vel.v[d] += f * latticeVelocity2D[i][d];
            }
        }
    }

    if constexpr (DIM == 3) {
        for (int i = 0; i < Q; ++i) {
            const T f = fin[i];
            for (int d = 0; d < DIM; ++d) {
                vel.v[d] += f * latticeVelocity3D[i][d];
            }
        }
    }

    for (int d = 0; d < DIM; ++d) {
        vel.v[d] /= rho;
    }
    return vel;
}

template <typename T, int DIM, int Q>
Neon::set::Container collide(Neon::domain::mGrid&                 grid,
                             T                                    omega0,
                             int                                  level,
                             int                                  max_level,
                             const Neon::domain::mGrid::Field<T>& fin,
                             Neon::domain::mGrid::Field<T>&       fout)
{
    return grid.getContainer(
        "Collide" + std::to_string(level), level,
        [=](Neon::set::Loader& loader) {
            const auto& in = fin.load(loader, level, Neon::MultiResCompute::MAP);
            auto        out = fout.load(loader, level, Neon::MultiResCompute::MAP);
            const T     omega = computeOmega(omega0, level, max_level);

            return [=] NEON_CUDA_HOST_DEVICE(const typename Neon::domain::bGrid::Cell& cell) mutable {
                constexpr auto t = latticeWeight<DIM, Q>();

                //fin
                T ins[Q];
                for (int i = 0; i < Q; ++i) {
                    ins[i] = in(cell, i);
                }

                //density
                T rho = 0;
                for (int i = 0; i < Q; ++i) {
                    rho += ins[i];
                }

                //velocity
                const Neon::Vec_3d<T> vel = velocity<T, DIM, Q>(ins, rho);


                const T usqr = (3.0 / 2.0) * (vel.x * vel.x + vel.y * vel.y + vel.z * vel.z);
                for (int i = 0; i < Q; ++i) {
                    T cu = 0;
                    for (int d = 0; d < DIM; ++d) {
                        cu += latticeVelocity2D[i][d] * vel.v[d];
                    }
                    //equilibrium
                    T feq = rho * t.t[i] * (1. + cu + 0.5 * cu * cu - usqr);

                    //collide
                    out(cell, i) = ins[i] - omega * (ins[i] - feq);
                }
            };
        });
}

template <typename T, int DIM, int Q>
Neon::set::Container stream(Neon::domain::mGrid&                 grid,
                            int                                  level,
                            const Neon::domain::mGrid::Field<T>& fpop_postcollision,
                            Neon::domain::mGrid::Field<T>&       fpop_poststreaming)
{
    //regular Streaming of the normal voxels at level L which are not interfaced with L+1 and L-1 levels.
    //This is "pull" stream

    return grid.getContainer(
        "Stream" + std::to_string(level), level,
        [=](Neon::set::Loader& loader) {
            const auto& fpost_col = fpop_postcollision.load(loader, level, Neon::MultiResCompute::STENCIL);
            auto        fpost_stm = fpop_poststreaming.load(loader, level, Neon::MultiResCompute::MAP);

            return [=] NEON_CUDA_HOST_DEVICE(const typename Neon::domain::bGrid::Cell& cell) mutable {
                //If this cell has children i.e., it is been refined, that we should not work on it
                //because this cell is only there to allow query and not to operate on
                if (!fpost_stm.hasChildren(cell)) {

                    for (int8_t q = 0; q < Q; ++q) {
                        Neon::int8_3d dir;
                        if constexpr (DIM == 2) {
                            dir = Neon::int8_3d(-latticeVelocity2D[q][0], -latticeVelocity2D[q][1], 0);
                        }
                        if constexpr (DIM == 3) {
                            dir = Neon::int8_3d(-latticeVelocity3D[q][0], -latticeVelocity3D[q][1], -latticeVelocity3D[q][2]);
                        }
                        //if the neighbor cell has children, then this 'cell' is interfacing with L-1 (fine) along q direction
                        if (!fpost_stm.hasChildren(cell, dir)) {
                            auto neighbor = fpost_col.nghVal(cell, dir, q, T(0));
                            if (neighbor.isValid) {
                                fpost_stm(cell, q) = neighbor.value;
                            }
                        }
                    }
                }
            };
        });
}

template <typename T, int DIM, int Q>
Neon::set::Container explosionPull(Neon::domain::mGrid&                 grid,
                                   int                                  level,
                                   const Neon::domain::mGrid::Field<T>& fpop_postcollision,
                                   Neon::domain::mGrid::Field<T>&       fpop_poststreaming)
{
    // Initiated by the fine level (hence "pull"), this function performs a coarse (level+1) to
    // fine (level) communication or "explosion" by simply distributing copies of coarse grid onto the fine grid.
    // In other words, this function updates the "halo" cells of the fine level by making copies of the coarse cell
    // values.


    return grid.getContainer(
        "explosionPull" + std::to_string(level), level,
        [=](Neon::set::Loader& loader) {
            const auto& fpost_col = fpop_postcollision.load(loader, level, Neon::MultiResCompute::STENCIL_UP);
            auto        fpost_stm = fpop_poststreaming.load(loader, level, Neon::MultiResCompute::MAP);

            return [=] NEON_CUDA_HOST_DEVICE(const typename Neon::domain::bGrid::Cell& cell) mutable {
                for (int8_t q = 0; q < Q; ++q) {
                    //If this cell has children i.e., it is been refined, that we should not work on it
                    //because this cell is only there to allow query and not to operate on
                    if (!fpost_stm.hasChildren(cell)) {
                        Neon::int8_3d dir;
                        if constexpr (DIM == 2) {
                            dir = Neon::int8_3d(-latticeVelocity2D[q][0], -latticeVelocity2D[q][1], 0);
                        }
                        if constexpr (DIM == 3) {
                            dir = Neon::int8_3d(-latticeVelocity3D[q][0], -latticeVelocity3D[q][1], -latticeVelocity3D[q][2]);
                        }

                        //if the neighbor cell has children, then this 'cell' is interfacing with L-1 (fine) along q direction
                        //we want to only work on cells that interface with L+1 (coarse) cell along q                        
                        if (!fpost_stm.hasChildren(cell, dir)) {

                            //try to query the cell along this direction (opposite of the population direction) as we do
                            //in 'normal' streaming
                            auto neighborCell = fpost_col.setNghCell(cell, dir);
                            if (!neighborCell.isActive()) {
                                //only if we can not do normal streaming, then we may have a coarser neighbor from which
                                //we can read this pop

                                //get the uncle direction/offset i.e., the neighbor of the cell's parent 
                                //this direction/offset is wrt to the cell's parent 
                                Neon::int8_3d uncleDir = explosionUnlceOffset(cell, dir);

                                auto uncle = fpost_col.uncleVal(cell, uncleDir, q, T(0));
                                if (uncle.isValid) {
                                    fpost_stm(cell, q) = uncle.value;
                                }
                            }
                        }
                    }
                }
            };
        });
}


template <typename T, int DIM, int Q>
Neon::set::Container coalescencePull(Neon::domain::mGrid&           grid,
                                     int                            level,
                                     Neon::domain::mGrid::Field<T>& fpop_poststreaming)
{
    //TODO
    // Initiated by the coarse level (hence "pull"), this function performs fine (level-1) to coarse
    // (level) communication or "coalescence" by simply averaging the fine data stored in self.fpop_halo

    return grid.getContainer(
        "coalescencePull" + std::to_string(level), level,
        [=](Neon::set::Loader& loader) {
            auto fpost_stm = fpop_poststreaming.load(loader, level, Neon::MultiResCompute::STENCIL_DOWN);

            return [=] NEON_CUDA_HOST_DEVICE(const typename Neon::domain::bGrid::Cell& cell) mutable {
                for (int i = 0; i < Q; ++i) {
                }
            };
        });
}

template <typename T, int DIM, int Q>
void stream(Neon::domain::mGrid&                 grid,
            int                                  level,
            const int                            max_level,
            const Neon::domain::mGrid::Field<T>& fpop_postcollision,
            Neon::domain::mGrid::Field<T>&       fpop_poststreaming,
            std::vector<Neon::set::Container>&   containers)
{
    containers.push_back(stream<T, DIM, Q>(grid, level, fpop_postcollision, fpop_poststreaming));

    /*
    * Streaming for interface voxels that have
    *  (i) coarser or (ii) finer neighbors at level+1 and level-1 and hence require
    *  (i) "explosion" or (ii) coalescence
    */
    if (level != max_level - 1) {
        /* Explosion: pull missing populations from coarser neighbors by copying coarse (level+1) to fine (level) 
        * neighbors, initiated by the fine level ("Pull").
        */
        containers.push_back(explosionPull<T, DIM, Q>(grid, level, fpop_postcollision, fpop_poststreaming));
    }

    if (level != 0) {
        /* Coalescence: pull missing populations from finer neighbors by "smart" averaging fine (level-1) 
        * to coarse (level) communication, initiated by the coarse level ("Pull").
        */
        containers.push_back(coalescencePull<T, DIM, Q>(grid, level, fpop_poststreaming));
    }
}

template <typename T, int DIM, int Q>
void nonUniformTimestepRecursive(Neon::domain::mGrid&               grid,
                                 const T                            omega0,
                                 const int                          level,
                                 const int                          max_level,
                                 Neon::domain::mGrid::Field<T>&     fin,
                                 Neon::domain::mGrid::Field<T>&     fout,
                                 std::vector<Neon::set::Container>& containers)
{
    // 1) collision for all voxels at level L=level
    containers.push_back(collide<T, DIM, Q>(grid, omega0, level, max_level, fin, fout));

    // 2) Storing fine(level) data for later "coalescence" pulled by the coarse(level)

    // 3) recurse down
    if (level != 0) {
        nonUniformTimestepRecursive<T, DIM, Q>(grid, omega0, level - 1, max_level, fin, fout, containers);
    }

    // 4) Streaming step that also performs the necessary "explosion" and "coalescence" steps.
    stream<T, DIM, Q>(grid, level, max_level, fout, fin, containers);

    // 5) stop
    if (level == max_level - 1) {
        return;
    }

    // 6) collision for all voxels at level L = level
    containers.push_back(collide<T, DIM, Q>(grid, omega0, level, max_level, fin, fout));

    // 7) Storing fine(level) data for later "coalescence" pulled by the coarse(level)

    // 8) recurse down
    if (level != 0) {
        nonUniformTimestepRecursive<T, DIM, Q>(grid, omega0, level - 1, max_level, fin, fout, containers);
    }

    // 9) Streaming step
    stream<T, DIM, Q>(grid, level, max_level, fout, fin, containers);
}


int main(int argc, char** argv)
{
    Neon::init();

    if (Neon::sys::globalSpace::gpuSysObjStorage.numDevs() > 0) {
        using T = double;

        //Neon grid
        auto             runtime = Neon::Runtime::openmp;
        std::vector<int> gpu_ids{0};
        Neon::Backend    backend(gpu_ids, runtime);

        constexpr int DIM = 2;
        constexpr int Q = (DIM == 2) ? 9 : 19;

        const int dim_x = 12;
        const int dim_y = 12;
        const int dim_z = (DIM < 3) ? 4 : 4;

        const Neon::index_3d grid_dim(dim_x, dim_y, dim_z);

        const Neon::domain::mGridDescriptor descriptor({1, 1, 1});


        Neon::domain::mGrid grid(
            backend, grid_dim,
            {[&](const Neon::index_3d id) -> bool {
                 return id.x > 7;
             },
             [&](const Neon::index_3d& id) -> bool {
                 return id.x > 3;
             },
             [&](const Neon::index_3d&) -> bool {
                 return true;
             }},
            create_stencil<DIM, Q>(), descriptor);

        //grid.topologyToVTK("lbm.vtk", false);


        //LBM problem
        const int max_iter = 300;
        const T   ulb = 0.01;
        const T   Re = 20;
        const T   clength = grid_dim.x;
        const T   visclb = ulb * clength / Re;
        const T   smagorinskyConstant = 0.02;
        const T   omega = 1.0 / (3. * visclb + 0.5);

        auto fin = grid.newField<T>("fin", Q, 0);
        auto fout = grid.newField<T>("fout", Q, 0);

        //TODO init fin and fout

        fin.updateCompute();
        fout.updateCompute();

        std::vector<Neon::set::Container> containers;

        nonUniformTimestepRecursive<T, DIM, Q>(grid,
                                               omega,
                                               descriptor.getDepth() - 1,
                                               descriptor.getDepth(),
                                               fin, fout, containers);

        Neon::skeleton::Skeleton skl(grid.getBackend());
        skl.sequence(containers, "MultiResLBM");
        //skl.ioToDot("MultiRes");

        skl.run();

        grid.getBackend().syncAll();
        fin.updateIO();
        fout.updateIO();
    }
}