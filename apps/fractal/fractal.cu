#include "hip/hip_runtime.h"
#include <iomanip>
#include <sstream>

#include "Neon/Neon.h"
#include "Neon/domain/dGrid.h"
#include "Neon/skeleton/Skeleton.h"

template <typename Field>
inline void draw_pixels(const int t, Field& field)
{
    printf("\n Exporting Frame =%d", t);
    int                precision = 4;
    std::ostringstream oss;
    oss << std::setw(precision) << std::setfill('0') << t;
    std::string fname = "frame_" + oss.str();
    field.ioToVtk(fname, "pixels");
}

NEON_CUDA_HOST_DEVICE inline Neon::float_2d complex_sqr(Neon::float_2d& z)
{
    return Neon::float_2d(z.x * z.x - z.y * z.y, z.x * z.y * 2.0f);
}

NEON_CUDA_HOST_DEVICE inline Neon::float_2d complex_pow(Neon::float_2d& z, Neon::float_1d& n)
{
    Neon::float_1d radius = pow(z.norm(), n);
    Neon::float_1d angle = n * atan2(z.y, z.x);
    return Neon::float_2d(radius * cos(angle), radius * sin(angle));
}

template <typename Field>
inline Neon::set::Container FractalsContainer(Field&  pixels,
                                              int32_t& time,
                                              int32_t  n)
{
    return pixels.getGrid().newContainer(
        "FractalContainer", [&, n](Neon::set::Loader& L) {
            auto& px = L.load(pixels);
            auto& t = time;

            return [=] NEON_CUDA_HOST_DEVICE(
                       const typename Field::Idx& idx) mutable {
                auto id = px.getGlobalIndex(idx);

                Neon::float_2d c(-0.8, cos(t * 0.03) * 0.2);
                Neon::float_2d z((float(id.x) / float(n)) - 1.0f,
                                 (float(id.y) / float(n)) - 0.5f);
                z *= 2.0f;
                float iterations = 0;
                while (z.norm() < 20 && iterations < 50) {
                    z = complex_sqr(z) + c;
                    iterations += 1;
                }
                px(idx, 0) = 1.0f - iterations * 0.02;
            };
        });
}

int main(int argc, char** argv)
{
    Neon::init();
    if ( Neon::Backend::countAvailableGpus() > 0) {
        int32_t          n = 320;
        Neon::index_3d   dim(2 * n, n, 1);
        std::vector<int> gpu_ids{0};

        auto runtime = Neon::Runtime::stream;

        //runtime = Neon::Runtime::openmp;

        Neon::Backend backend(gpu_ids, runtime);

        using Grid = Neon::dGrid;
        Grid grid(
            backend, dim,
            [](const Neon::index_3d& idx) -> bool { return true; },
            Neon::domain::Stencil::s7_Laplace_t());

        int   cardinality = 1;
        float inactiveValue = 0.0f;
        auto  pixels = grid.template newField<float>("pixels", cardinality, inactiveValue);

        Neon::skeleton::Skeleton skeleton(backend);

        int32_t time;
        skeleton.sequence({FractalsContainer(pixels, time, n)}, "fractal");


        for (time = 0; time < 1000; ++time) {
            skeleton.run();

            pixels.updateHostData(0);
            draw_pixels(time, pixels);
        }
    }
}